#include "hip/hip_runtime.h"
/*
 MIT License

 Copyright (c) 2024 Carlos Cabaço Tojal

 Permission is hereby granted, free of charge, to any person obtaining a copy
 of this software and associated documentation files (the "Software"), to deal
 in the Software without restriction, including without limitation the rights
 to use, copy, modify, merge, publish, distribute, sublicense, and/or sell
 copies of the Software, and to permit persons to whom the Software is
 furnished to do so, subject to the following conditions:

 The above copyright notice and this permission notice shall be included in all
 copies or substantial portions of the Software.

 THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT. IN NO EVENT SHALL THE
 AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING FROM,
 OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER DEALINGS IN THE
 SOFTWARE.

 */

#include <pcl_aggregator_core/cuda/CUDAPointClouds.h>

namespace pcl_aggregator {
    namespace cuda {
        namespace pointclouds {

            static __host__ int setPointCloudLabelCuda(const pcl::PointCloud<pcl::PointXYZRGBL>::Ptr& cloud, std::uint32_t label) {
                hipError_t err = hipSuccess;
                hipStream_t stream;

                // declare the device input point array
                pcl::PointXYZRGBL *d_cloud;

                if((err = hipSetDevice(0)) != hipSuccess) {
                    std::cerr << "Error setting the CUDA device: " << hipGetErrorString(err) << std::endl;
                    return -1;
                }

                // create a stream
                if ((err = hipStreamCreate(&stream)) != hipSuccess) {
                    std::cerr << "Error creating the label-setting CUDA stream: " << hipGetErrorString(err) << std::endl;
                    return -2;
                }

                // allocate memory on the device to store the input pointcloud
                if ((err = hipMalloc(&d_cloud, cloud->size() * sizeof(pcl::PointXYZRGBL))) != hipSuccess) {
                    std::cerr << "Error allocating memory for the pointcloud: " << hipGetErrorString(err) << std::endl;
                    return -3;
                }

                // copy the input pointcloud to the device
                if ((err = hipMemcpy(d_cloud, cloud->points.data(), cloud->size() * sizeof(pcl::PointXYZRGBL),
                                      hipMemcpyHostToDevice)) != hipSuccess) {
                    std::cerr << "Error copying the input pointcloud to the device (set label): " << hipGetErrorString(err)
                              << std::endl;
                    return -4;
                }

                // call the kernel
                dim3 block(512);
                dim3 grid((cloud->size() + block.x - 1) / block.x);
                setPointLabelKernel<<<grid, block, 0, stream>>>(d_cloud, label, cloud->size());

                // wait for the stream
                if ((err = hipStreamSynchronize(stream)) != hipSuccess) {
                    std::cerr << "Error waiting for the label-setting stream: " << hipGetErrorString(err) << std::endl;
                    return -5;
                }

                // copy the output pointcloud back to the host
                if ((err = hipMemcpy(cloud->points.data(), d_cloud, cloud->size() * sizeof(pcl::PointXYZRGBL),
                                      hipMemcpyDeviceToHost)) != hipSuccess) {
                    std::cerr << "Error copying the output pointcloud to the host (labelling): " << hipGetErrorString(err)
                              << std::endl;
                    return -6;
                }

                // free the memory
                if ((err = hipFree(d_cloud)) != hipSuccess) {
                    std::cerr << "Error freeing the pointcloud from device memory: " << hipGetErrorString(err)
                              << std::endl;
                    return -7;
                }

                // destroy the stream
                if ((err = hipStreamDestroy(stream)) != hipSuccess) {
                    std::cerr << "Error destroying the CUDA stream: " << hipGetErrorString(err) << std::endl;
                    return -8;
                }

                return 0;
            }

            static __global__ void setPointLabelKernel(pcl::PointXYZRGBL *points, std::uint32_t label, int num_points) {
                std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                if (idx < num_points) {
                    points[idx].label = label;
                }
            }

            static __host__ int transformPointCloudCuda(const pcl::PointCloud<pcl::PointXYZRGBL>::Ptr& cloud, const Eigen::Affine3d& tf) {

                hipError_t err = hipSuccess;
                hipStream_t stream;

                if((err = hipSetDevice(0)) != hipSuccess) {
                    std::cerr << "Error setting the CUDA device: " << hipGetErrorString(err) << std::endl;
                    return -1;
                }

                if ((err = hipStreamCreate(&stream)) != hipSuccess) {
                    std::cerr << "Error creating pointcloud transform stream: " << hipGetErrorString(err) << std::endl;
                    return -2;
                }

                // allocate device memory for the pointcloud
                pcl::PointXYZRGBL *d_cloud;
                if ((err = hipMalloc(&d_cloud, cloud->size() * sizeof(pcl::PointXYZRGBL))) != hipSuccess) {
                    std::cerr << "Error allocating memory for the pointcloud: " << hipGetErrorString(err) << std::endl;
                    return -3;
                }

                // copy the pointcloud to the device
                if ((err = hipMemcpy(d_cloud, cloud->points.data(), cloud->size() * sizeof(pcl::PointXYZRGBL),
                                      hipMemcpyHostToDevice)) != hipSuccess) {
                    std::cerr << "Error copying the input pointcloud to the device (transform): " << hipGetErrorString(err)
                              << std::endl;
                    return -4;
                }

                // call the kernel
                dim3 block(512);
                dim3 grid((cloud->size() + block.x - 1) / block.x);
                transformPointKernel<<<grid, block, 0, stream>>>(d_cloud, tf.matrix(), cloud->size());

                // wait for the stream
                if ((err = hipStreamSynchronize(stream)) != hipSuccess) {
                    std::cerr << "Error waiting for the transform stream: " << hipGetErrorString(err) << std::endl;
                    return -5;
                }

                // copy the output pointcloud back to the host
                if ((err = hipMemcpy(cloud->points.data(), d_cloud, cloud->size() * sizeof(pcl::PointXYZRGBL),
                                      hipMemcpyDeviceToHost)) != hipSuccess) {
                    std::cerr << "Error copying the output pointcloud to the host (transform): " << hipGetErrorString(err)
                              << std::endl;
                    return -6;
                }

                // free the memory
                if ((err = hipFree(d_cloud)) != hipSuccess) {
                    std::cerr << "Error freeing the pointcloud from device memory: " << hipGetErrorString(err)
                              << std::endl;
                    return -7;
                }

                // destroy the stream
                if ((err = hipStreamDestroy(stream)) != hipSuccess) {
                    std::cerr << "Error destroying the CUDA stream: " << hipGetErrorString(err) << std::endl;
                    return -8;
                }

                return 0;
            }

            static __global__ void transformPointKernel(pcl::PointXYZRGBL *points, Eigen::Matrix4d transform, int num_points) {
                std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                if (idx < num_points) {
                    Eigen::Vector4d p(points[idx].x, points[idx].y, points[idx].z, 1.0f);
                    p = transform * p;
                    points[idx].x = p(0);
                    points[idx].y = p(1);
                    points[idx].z = p(2);
                }
            }

            static __host__ int concatenatePointCloudsCuda(const pcl::PointCloud<pcl::PointXYZRGBL>::Ptr& cloud1,
                                                     const pcl::PointCloud<pcl::PointXYZRGBL>& cloud2) {

                hipError_t err = hipSuccess;
                hipStream_t stream;

                if((err = hipSetDevice(0)) != hipSuccess) {
                    std::cerr << "Error setting the CUDA device: " << hipGetErrorString(err) << std::endl;
                    return -1;
                }

                // create a stream
                if ((err = hipStreamCreate(&stream)) != hipSuccess) {
                    std::cerr << "Error creating pointcloud concatenation stream: " << hipGetErrorString(err) << std::endl;
                    return -2;
                }

                // resize the cloud1
                std::size_t cloud1OriginalSize = cloud1->size();
                std::size_t cloud1NewSize = cloud1OriginalSize + cloud2.size();
                cloud1->resize(cloud1NewSize);

                // allocate cloud1 on the device - allocate with sufficient space to the concatenation
                pcl::PointXYZRGBL *d_cloud1;
                if ((err = hipMalloc(&d_cloud1, cloud1NewSize * sizeof(pcl::PointXYZRGBL))) != hipSuccess) {
                    std::cerr << "Error allocating memory for cloud1: " << hipGetErrorString(err) << std::endl;
                    return -3;
                }

                // copy cloud1 to the device
                if ((err = hipMemcpy(d_cloud1, cloud1->points.data(), cloud1NewSize * sizeof(pcl::PointXYZRGBL),
                                      hipMemcpyHostToDevice)) != hipSuccess) {
                    std::cerr << "Error copying cloud1 to the device: " << hipGetErrorString(err)
                              << std::endl;
                    return -4;
                }

                // allocate cloud2 on the device
                pcl::PointXYZRGBL *d_cloud2;
                if((err = hipMalloc(&d_cloud2, cloud2.size() * sizeof(pcl::PointXYZRGBL))) != hipSuccess) {
                    std::cerr << "Error allocating memory for cloud2: " << hipGetErrorString(err) << std::endl;
                    return -5;
                }

                // copy cloud2 to the device
                if((err = hipMemcpy(d_cloud2, cloud2.points.data(), cloud2.size() * sizeof(pcl::PointXYZRGBL),
                                     hipMemcpyHostToDevice)) != hipSuccess) {
                    std::cerr << "Error copying cloud2 to the device: " << hipGetErrorString(err) << std::endl;
                    return -6;
                }

                // call the kernel
                dim3 block(512);
                // will be needed as much thread as the size of the cloud2, ideally
                dim3 grid((cloud2.size() + block.x - 1) / block.x);
                concatenatePointCloudsKernel<<<grid, block, 0, stream>>>(d_cloud1,
                                                                         cloud1OriginalSize, d_cloud2,
                                                                         cloud2.size());

                // wait for the stream to synchronize the threads
                if ((err = hipStreamSynchronize(stream)) != hipSuccess) {
                    std::cerr << "Error waiting for the concatenation stream: " << hipGetErrorString(err) << std::endl;
                    return -7;
                }

                // copy cloud1 back to the host
                if ((err = hipMemcpy(cloud1->points.data(), d_cloud1, cloud1NewSize * sizeof(pcl::PointXYZRGBL),
                                      hipMemcpyDeviceToHost)) != hipSuccess) {
                    std::cerr << "Error copying cloud1 to the host: " << hipGetErrorString(err)
                              << std::endl;
                    return -8;
                }

                // free cloud1
                if ((err = hipFree(d_cloud1)) != hipSuccess) {
                    std::cerr << "Error freeing cloud1 from device memory: " << hipGetErrorString(err)
                              << std::endl;
                    return -9;
                }

                // free cloud2
                if ((err = hipFree(d_cloud2)) != hipSuccess) {
                    std::cerr << "Error freeing cloud2 from device memory: " << hipGetErrorString(err)
                              << std::endl;
                    return -10;
                }

                // destroy the stream
                if ((err = hipStreamDestroy(stream)) != hipSuccess) {
                    std::cerr << "Error destroying the CUDA stream: " << hipGetErrorString(err) << std::endl;
                    return -11;
                }

                return 0;

            }

            static __global__ void concatenatePointCloudsKernel(pcl::PointXYZRGBL* cloud1, std::size_t cloud1_original_size,
                                                         pcl::PointXYZRGBL* cloud2, std::size_t cloud2_size) {
                // calculate the index
                std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

                // check boundaries - should range between 0 and cloud2_size
                if(idx >= cloud2_size)
                    return;

                // copy the point from cloud2 to cloud1
                cloud1[cloud1_original_size+idx] = cloud2[idx];
            }
        }
    } // pcl_aggregator
} // cuda