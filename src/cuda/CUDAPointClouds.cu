#include "hip/hip_runtime.h"
//
// Created by carlostojal on 30-04-2023.
//

#include <pcl_aggregator_core/cuda/CUDAPointClouds.cuh>

namespace pcl_aggregator {
    namespace cuda {
        namespace pointclouds {

            __host__ void setPointCloudLabelCuda(const pcl::PointCloud<pcl::PointXYZRGBL>::Ptr& cloud, std::uint32_t label) {
                hipError_t err = hipSuccess;
                hipStream_t stream;

                // declare the device input point array
                pcl::PointXYZRGBL *d_cloud;

                if((err = hipSetDevice(0)) != hipSuccess) {
                    std::cerr << "Error setting the CUDA device: " << hipGetErrorString(err) << std::endl;
                    return;
                }

                // create a stream
                if ((err = hipStreamCreate(&stream)) != hipSuccess) {
                    std::cerr << "Error creating the label-setting CUDA stream: " << hipGetErrorString(err) << std::endl;
                    return;
                }

                // allocate memory on the device to store the input pointcloud
                if ((err = hipMalloc(&d_cloud, cloud->size() * sizeof(pcl::PointXYZRGBL))) != hipSuccess) {
                    std::cerr << "Error allocating memory for the pointcloud: " << hipGetErrorString(err) << std::endl;
                    return;
                }

                // copy the input pointcloud to the device
                if ((err = hipMemcpy(d_cloud, cloud->points.data(), cloud->size() * sizeof(pcl::PointXYZRGBL),
                                      hipMemcpyHostToDevice)) != hipSuccess) {
                    std::cerr << "Error copying the input pointcloud to the device (set label): " << hipGetErrorString(err)
                              << std::endl;
                    return;
                }

                // call the kernel
                dim3 block(512);
                dim3 grid((cloud->size() + block.x - 1) / block.x);
                setPointLabelKernel<<<grid, block, 0, stream>>>(d_cloud, label, cloud->size());

                // wait for the stream
                if ((err = hipStreamSynchronize(stream)) != hipSuccess) {
                    std::cerr << "Error waiting for the label-setting stream: " << hipGetErrorString(err) << std::endl;
                    return;
                }

                // copy the output pointcloud back to the host
                if ((err = hipMemcpy(cloud->points.data(), d_cloud, cloud->size() * sizeof(pcl::PointXYZRGBL),
                                      hipMemcpyDeviceToHost)) != hipSuccess) {
                    std::cerr << "Error copying the output pointcloud to the host (labelling): " << hipGetErrorString(err)
                              << std::endl;
                    return;
                }

                // free the memory
                if ((err = hipFree(d_cloud)) != hipSuccess) {
                    std::cerr << "Error freeing the pointcloud from device memory: " << hipGetErrorString(err)
                              << std::endl;
                    return;
                }

                // destroy the stream
                if ((err = hipStreamDestroy(stream)) != hipSuccess) {
                    std::cerr << "Error destroying the CUDA stream: " << hipGetErrorString(err) << std::endl;
                    return;
                }
            }

            __global__ void setPointLabelKernel(pcl::PointXYZRGBL *points, std::uint32_t label, int num_points) {
                std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                if (idx < num_points) {
                    points[idx].label = label;
                }
            }

            __host__ void transformPointCloudCuda(const pcl::PointCloud<pcl::PointXYZRGBL>::Ptr& cloud, const Eigen::Affine3d& tf) {

                hipError_t err = hipSuccess;
                hipStream_t stream;

                if((err = hipSetDevice(0)) != hipSuccess) {
                    std::cerr << "Error setting the CUDA device: " << hipGetErrorString(err) << std::endl;
                    return;
                }

                if ((err = hipStreamCreate(&stream)) != hipSuccess) {
                    std::cerr << "Error creating pointcloud transform stream: " << hipGetErrorString(err) << std::endl;
                    return;
                }

                // allocate device memory for the pointcloud
                pcl::PointXYZRGBL *d_cloud;
                if ((err = hipMalloc(&d_cloud, cloud->size() * sizeof(pcl::PointXYZRGBL))) != hipSuccess) {
                    std::cerr << "Error allocating memory for the pointcloud: " << hipGetErrorString(err) << std::endl;
                    return;
                }

                // copy the pointcloud to the device
                if ((err = hipMemcpy(d_cloud, cloud->points.data(), cloud->size() * sizeof(pcl::PointXYZRGBL),
                                      hipMemcpyHostToDevice)) != hipSuccess) {
                    std::cerr << "Error copying the input pointcloud to the device (transform): " << hipGetErrorString(err)
                              << std::endl;
                    return;
                }

                // call the kernel
                dim3 block(512);
                dim3 grid((cloud->size() + block.x - 1) / block.x);
                transformPointKernel<<<grid, block, 0, stream>>>(d_cloud, tf.matrix(), cloud->size());

                // wait for the stream
                if ((err = hipStreamSynchronize(stream)) != hipSuccess) {
                    std::cerr << "Error waiting for the transform stream: " << hipGetErrorString(err) << std::endl;
                    return;
                }

                // copy the output pointcloud back to the host
                if ((err = hipMemcpy(cloud->points.data(), d_cloud, cloud->size() * sizeof(pcl::PointXYZRGBL),
                                      hipMemcpyDeviceToHost)) != hipSuccess) {
                    std::cerr << "Error copying the output pointcloud to the host (transform): " << hipGetErrorString(err)
                              << std::endl;
                    return;
                }

                // free the memory
                if ((err = hipFree(d_cloud)) != hipSuccess) {
                    std::cerr << "Error freeing the pointcloud from device memory: " << hipGetErrorString(err)
                              << std::endl;
                    return;
                }

                // destroy the stream
                if ((err = hipStreamDestroy(stream)) != hipSuccess) {
                    std::cerr << "Error destroying the CUDA stream: " << hipGetErrorString(err) << std::endl;
                    return;
                }
            }

            __global__ void transformPointKernel(pcl::PointXYZRGBL *points, Eigen::Matrix4d transform, int num_points) {
                std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;
                if (idx < num_points) {
                    Eigen::Vector4d p(points[idx].x, points[idx].y, points[idx].z, 1.0f);
                    p = transform * p;
                    points[idx].x = p(0);
                    points[idx].y = p(1);
                    points[idx].z = p(2);
                }
            }

            __host__ int concatenatePointCloudsCuda(const pcl::PointCloud<pcl::PointXYZRGBL>::Ptr& cloud1,
                                                     const pcl::PointCloud<pcl::PointXYZRGBL>& cloud2) {

                hipError_t err = hipSuccess;
                hipStream_t stream;

                if((err = hipSetDevice(0)) != hipSuccess) {
                    std::cerr << "Error setting the CUDA device: " << hipGetErrorString(err) << std::endl;
                    return -1;
                }

                // create a stream
                if ((err = hipStreamCreate(&stream)) != hipSuccess) {
                    std::cerr << "Error creating pointcloud concatenation stream: " << hipGetErrorString(err) << std::endl;
                    return -2;
                }

                // resize the cloud1
                std::size_t cloud1OriginalSize = cloud1->size();
                std::size_t cloud1NewSize = cloud1OriginalSize + cloud2.size();
                cloud1->resize(cloud1NewSize);

                // allocate cloud1 on the device - allocate with sufficient space to the concatenation
                pcl::PointXYZRGBL *d_cloud1;
                if ((err = hipMalloc(&d_cloud1, cloud1NewSize * sizeof(pcl::PointXYZRGBL))) != hipSuccess) {
                    std::cerr << "Error allocating memory for cloud1: " << hipGetErrorString(err) << std::endl;
                    return -3;
                }

                // copy cloud1 to the device
                if ((err = hipMemcpy(d_cloud1, cloud1->points.data(), cloud1NewSize * sizeof(pcl::PointXYZRGBL),
                                      hipMemcpyHostToDevice)) != hipSuccess) {
                    std::cerr << "Error copying cloud1 to the device: " << hipGetErrorString(err)
                              << std::endl;
                    return -4;
                }

                // allocate cloud2 on the device
                pcl::PointXYZRGBL *d_cloud2;
                if((err = hipMalloc(&d_cloud2, cloud2.size() * sizeof(pcl::PointXYZRGBL))) != hipSuccess) {
                    std::cerr << "Error allocating memory for cloud2: " << hipGetErrorString(err) << std::endl;
                    return -5;
                }

                // copy cloud2 to the device
                if((err = hipMemcpy(d_cloud2, cloud2.points.data(), cloud2.size() * sizeof(pcl::PointXYZRGBL),
                                     hipMemcpyHostToDevice)) != hipSuccess) {
                    std::cerr << "Error copying cloud2 to the device: " << hipGetErrorString(err) << std::endl;
                    return -6;
                }

                // call the kernel
                dim3 block(512);
                // will be needed as much thread as the size of the cloud2, ideally
                dim3 grid((cloud2.size() + block.x - 1) / block.x);
                concatenatePointCloudsKernel<<<grid, block, 0, stream>>>(d_cloud1,
                                                                         cloud1OriginalSize, d_cloud2,
                                                                         cloud2.size());

                // wait for the stream to synchronize the threads
                if ((err = hipStreamSynchronize(stream)) != hipSuccess) {
                    std::cerr << "Error waiting for the concatenation stream: " << hipGetErrorString(err) << std::endl;
                    return -7;
                }

                // copy cloud1 back to the host
                if ((err = hipMemcpy(cloud1->points.data(), d_cloud1, cloud1NewSize * sizeof(pcl::PointXYZRGBL),
                                      hipMemcpyDeviceToHost)) != hipSuccess) {
                    std::cerr << "Error copying cloud1 to the host: " << hipGetErrorString(err)
                              << std::endl;
                    return -8;
                }

                // free cloud1
                if ((err = hipFree(d_cloud1)) != hipSuccess) {
                    std::cerr << "Error freeing cloud1 from device memory: " << hipGetErrorString(err)
                              << std::endl;
                    return -9;
                }

                // free cloud2
                if ((err = hipFree(d_cloud2)) != hipSuccess) {
                    std::cerr << "Error freeing cloud2 from device memory: " << hipGetErrorString(err)
                              << std::endl;
                    return -10;
                }

                // destroy the stream
                if ((err = hipStreamDestroy(stream)) != hipSuccess) {
                    std::cerr << "Error destroying the CUDA stream: " << hipGetErrorString(err) << std::endl;
                    return -11;
                }

                return 0;

            }

            __global__ void concatenatePointCloudsKernel(pcl::PointXYZRGBL* cloud1, std::size_t cloud1_original_size,
                                                         pcl::PointXYZRGBL* cloud2, std::size_t cloud2_size) {
                // calculate the index
                std::size_t idx = blockIdx.x * blockDim.x + threadIdx.x;

                // check boundaries - should range between 0 and cloud2_size
                if(idx >= cloud2_size)
                    return;

                // copy the point from cloud2 to cloud1
                cloud1[cloud1_original_size+idx] = cloud2[idx];
            }
        }
    } // pcl_aggregator
} // cuda